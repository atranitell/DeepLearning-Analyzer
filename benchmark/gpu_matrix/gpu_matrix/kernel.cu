
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <random>
#include <iostream>
#include <time.h>


#define CUDA_CALL(x) { const hipError_t cudaError_code = (x) ; \
	if ( cudaError_code != hipSuccess) { \
		printf("\nCUDA Error: %s (err_num=%d)\n", hipGetErrorString(cudaError_code), cudaError_code); \
		hipDeviceReset(); assert(0); } }

#define CUDA_NUM_THREADS 1024
#define CUDA_NUM_BLOCKS 65535

#define CUDA_GET_BLOCKS(n) (((n)+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS)

#define KERNEL_CALL(x, n) x <<< CUDA_GET_BLOCKS(n), CUDA_NUM_THREADS >>>

#define CUDA_IDX (threadIdx.x + blockIdx.x * blockDim.x)

/*---------------------------------------------------------*/
//  SETTING
//
/*---------------------------------------------------------*/

#define data_type float
#define LENGTH (100)

/*---------------------------------------------------------*/
//  TEST GPU ADD
//  RESULT:
//
/*---------------------------------------------------------*/
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

void benchmark() {

	auto rand_clock_t = clock();

	// generate
	auto host_a = new data_type[LENGTH];
	auto host_b = new data_type[LENGTH];
	auto host_c = new data_type[LENGTH];

	// random
	srand(clock() - rand_clock_t);
	for (int i = 0; i < LENGTH; i++) {
		host_a[i] = data_type(rand()) / RAND_MAX;
		host_b[i] = data_type(rand()) / RAND_MAX;
		host_c[i] = data_type(rand()) / RAND_MAX;
	}

	// destory
	delete[] host_a;
	delete[] host_b;
	delete[] host_c;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
